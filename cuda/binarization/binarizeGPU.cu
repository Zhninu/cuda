#include "stdafx.h"
#include "binarizeGPU.cuh"
#include "binarizeKernel.cuh"
#include "../common/common.h" 

#define BLOCK_SIZE 256

int CBinarizeGPU::prepare(SDS3D *pVolumeData) 
{
	int nErr = EC_OK; 

	if (!pVolumeData) 
		return EC_ERR;

	int nVolSize = pVolumeData->nWid * pVolumeData->nHei * pVolumeData->nNum;
	int nBytes = nVolSize * sizeof(short);

	CheckCudaErrors(hipMalloc((void**)&m_dpVolume, nBytes));
	CheckCudaErrors(hipMalloc((void**)&m_dpBinarize, nBytes));
	CheckCudaErrors(hipMemcpyAsync(m_dpVolume, pVolumeData->pVolumeData, nBytes, hipMemcpyHostToDevice));

	return nErr;
}

int CBinarizeGPU::run(SDS3D& stBinarizeData)
{
	int nErr = EC_OK;

	int nSize = stBinarizeData.nWid * stBinarizeData.nHei * stBinarizeData.nNum;
	int nBytes = nSize * sizeof(short);

	dim3 block(BLOCK_SIZE);
	dim3 grid((stBinarizeData.nWid + BLOCK_SIZE - 1) / BLOCK_SIZE, stBinarizeData.nHei, stBinarizeData.nNum);
	binarizeKernel << <grid, block >> >(m_dpVolume, m_dpBinarize, nSize);
	CheckCudaErrors(hipMemcpyAsync(stBinarizeData.pVolumeData, m_dpBinarize, nBytes, hipMemcpyDeviceToHost));
	CheckCudaErrors(hipStreamSynchronize(hipStreamPerThread));

	return nErr;
}

int CBinarizeGPU::release() 
{
	int nErr = EC_OK;

	CheckCudaErrors(hipFree(m_dpVolume));
	CheckCudaErrors(hipFree(m_dpBinarize));

	return nErr;
}